#include "hip/hip_runtime.h"
/**************************************************************************************************************************************************
|       HBFT algorithm using Dynamic Programming in CUDA with Async memcopy
|       Author : Dinali Rosemin Dabarera
|       University of Peradeniya (EFac 2016) All Rights Reserved
|*************************************************************************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include "helpers.cuh"

/*
* Edge structure
*/
struct Edge {

    int from ;
    int to;
};


/*
* GPU Kernel to check whether the level array is filled or not
*/
__global__ void isLevelFilled(int * level, int * vertices, int * lev) {

    int j = (blockDim.x * blockIdx.x ) + threadIdx.x;

    if(level[j]==-1 && *lev==0) {
        *lev=1;

    }

}

/*
* GPU Kernel to update the level array of each vertex
*/
__global__ void BreadthFirstSearch( struct Edge * adjacencyList, int * vertices, int * level, int * lev, int * edges ) {

    int tid = (blockDim.x * blockIdx.x ) + threadIdx.x;
    *lev = 0;
    if(tid<*edges) {

        struct Edge element = adjacencyList[tid];
        if (level[element.from]>=0 and level[element.to]==-1) {
            level[element.to] = level[element.from]+1;
        }

    }


}

/*
* Main GPU Kernel which call other kernels : Dynamic programming. 
*/
__global__ void parentKenel(struct Edge * adjacencyList, int * vertices, int * level, int * lev, int * edges) {
   
 *lev=1;

    while(*lev==1) {
	/*
         * Update level array
         */
        BreadthFirstSearch<<<ceil(*edges/256.0),256>>> (adjacencyList,vertices,level,lev,edges);
        hipDeviceSynchronize();
       /*
        * Check level array
        */
        isLevelFilled<<<ceil(*vertices/256.0),256>>>(level,vertices,lev);
        hipDeviceSynchronize();
    }



}

int max_array(int a[], int num_elements) {
    int i, max=-1;
    for (i=0; i<num_elements; i++) {
        if (a[i]>max) {
            max=a[i];
        }
    }
    return(max);
}



/*
* Main Program starts here. 
*/


int main(int arg,char** args) {

    
   //hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    
    int device =0;
    
    /*
     * Select the GPU card: For Dynamic programing: GPU over 3.5 architecture
     */
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    hipSetDevice(device);
    fprintf(stderr,"Device name: %s\n", prop.name);

    
    /*
     * Start Clock
     */
    hipEvent_t start,stop;
    float elapsedtime;
    hipEventCreate(&start);
    hipEventRecord(start,0);





    int noOfRows;
    int   i;
    int v1,v2;
    int finalLevel;

    /*
     * Host variables
     */
    int * Hvertices=(int *) malloc(sizeof(int));
    int * Hedges=(int *) malloc(sizeof(int));
    int * Hlev =(int *)malloc(sizeof(int));
    int * HstartArrayCount = (int *) malloc(sizeof(int));
	

    /*
     * Read data from Graph file
     */
    FILE* fileNew = fopen(args[1], "r");

    fscanf(fileNew, "%d",&finalLevel);
    fscanf(fileNew, "%d %d %d",&noOfRows, Hvertices, Hedges);
  

    int * Hlevel= (int *)malloc(sizeof(int)*(*Hvertices));
    struct Edge * HedgeList =(struct Edge * )malloc(sizeof(struct Edge)*(*Hedges));


    *Hlev = 0;

    for (i = 0; i < *Hvertices; ++i) {

        Hlevel[i] = -1;
    }

    int val;

    for (i = 0; i < *Hedges; ++i) {

        fscanf(fileNew, "%d %d %d",&v1, &v2, &val);

        // Adding edge v1 --> v2
        HedgeList[i].from = v1;
        HedgeList[i].to = v2;

    }

 
    /*
     * Read data from Input vertex file
     */

    FILE * vectorFile= fopen(args[2],"r");
    fscanf(vectorFile,"%d",HstartArrayCount);


    int tempVal;
    for(i=0; i<*HstartArrayCount; i++) {

        fscanf(vectorFile,"%d",&tempVal);

        Hlevel[tempVal]=0;

    }

    /*
     * Device variables
     */

    int * Dvertices;
    int * Dedges;
    int * Dlev ;
    int * DstartArrayCount ;
    int * Dlevel;
    struct Edge * DedgeList ;

    /*
     * Allocate memory on Device
     */
    checkCuda(hipMalloc((void **)&Dvertices,sizeof(int)));
    checkCuda(hipMalloc((void **)&Dedges,sizeof(int)));
    checkCuda(hipMalloc((void **)&Dlev,sizeof(int)));
    checkCuda(hipMalloc((void **)&DstartArrayCount,sizeof(int)));
    checkCuda(hipMalloc((void **)&Dlevel,sizeof(int)*(*Hvertices)));
    checkCuda(hipMalloc((void **)&DedgeList,sizeof(struct Edge)* (*Hedges)));

    /*
     * Copy data from Host to Device
     */
    checkCuda(hipMemcpyAsync(Dvertices,Hvertices,sizeof(int),hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(Dedges,Hedges,sizeof(int),hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(Dlev,Hlev,sizeof(int),hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(DstartArrayCount,HstartArrayCount,sizeof(int),hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(Dlevel,Hlevel,sizeof(int)*(*Hvertices),hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(DedgeList,HedgeList,sizeof(struct Edge)*(*Hedges),hipMemcpyHostToDevice));


   /*
    * Main kernel call
    */
    parentKenel<<<1,1>>>(DedgeList,Dvertices,Dlevel,Dlev,Dedges);
    hipDeviceSynchronize();
    checkCudaError();
    checkCuda(hipMemcpy(Hlevel,Dlevel,sizeof(int)*(*Hvertices),hipMemcpyDeviceToHost));


    /*
     * Free memory on the Device
     */
    hipFree(Dvertices);
    hipFree(Dedges);
    hipFree(Dlev);
    hipFree(DstartArrayCount );
    hipFree(Dlevel);
    hipFree(DedgeList);

    /*
     * Stop the Clock
     */
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedtime,start,stop);

    printf("%d, %d, %.8f \n",*Hvertices, *Hedges,elapsedtime/(float)1000);

    /*
     * Print vertices according to the level order
     */
      	 printf("\nLevel and Parent Arrays -\n");
               for (i = 0; i < *Hvertices; ++i) {
                    printf("Level of Vertex %d is %d\n",
                                              i, Hlevel[i]);
                }

                printf("vertices in level order when traversing :\n");

                int b;
                 for(b=0;b<=max_array(Hlevel,*Hvertices);b++){
                   for (i = 0; i < *Hvertices; ++i) {
                       if(Hlevel[i]==b){
                            printf("%d ,", i);
                       }

                   }
                    printf("  |  ");
                 }
 
    /*
     * Free Host memory
     */
    free(Hvertices);
    free(Hedges);
    free(Hlev);
    free(HstartArrayCount );
    free(Hlevel);
    free(HedgeList);


    return 0;
}
